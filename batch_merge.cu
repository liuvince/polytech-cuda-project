
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define X 0
#define Y 1 

bool isSumOfLengthEqualtoD(int *, int *, int, int);
bool isPowerOfTwo(int);

__global__ void mergeSmallBatch_k(int * A, int * sizeA, int *B, int *sizeB, int *M, const int d, int N){

	// Question 4 : Indices importantes
	const int tidx = threadIdx.x % d;				// Numéro de la diagonal dans le tableau numéro Qt
	const int Qt = (threadIdx.x - tidx) / d;			// Numéro du tableau par rapport au tableau shared
	const int gbx = Qt + blockIdx.x * (blockDim.x / d);		// Numéro du tableau par rapport au tableau global

	const int sizeAi = sizeA[gbx];				// Taille du tableau considéré
	const int sizeBi = sizeB[gbx];				// Taille du tableau considéré

	__shared__ int Atemp[1024];				// Tableau partagé par les threads d'un bloc
	__shared__ int Btemp[1024];				// Tableau partagé par les threads d'un bloc

	Atemp[Qt * d + tidx] = A[gbx * d + tidx];		// Remplissage du tableau
	Btemp[Qt * d + tidx] = B[gbx * d + tidx];		// Remplissage du tableau

	__syncthreads();					// Attente de la synchronisation de tous les threads du bloc
//	printf("blockId.x = %d | threadIdx. x = %d | tidx = %d | Qt = %d | gbx = %d\n", blockIdx.x, threadIdx.x, tidx, Qt, gbx);

	if (gbx * d + tidx >= N * d){
		return;
	}
	// ======================== //
	//        Merge path        //
	// ======================== //
	int K[2];
	int P[2];

	if (tidx > sizeAi) {
		K[X] = tidx - sizeAi;
		K[Y] = sizeAi;
		P[X] = sizeAi;
		P[Y] = tidx - sizeAi;
	}
	else {
		K[X] = 0;
		K[Y] = tidx;
		P[X] = tidx;
		P[Y] = 0;
	}

	while (1) {
		int offset = (abs(K[Y] - P[Y]))/2;
		int Q[2] = {K[X] + offset, K[Y] - offset};

		if (Q[Y] >= 0 && Q[X] <= sizeBi && (Q[Y] == sizeAi || Q[X] == 0 || Atemp[Qt*d + Q[Y]] > Btemp[Qt*d + Q[X]-1])) {
			if (Q[X] == sizeBi || Q[Y] == 0 || Atemp[Qt*d + Q[Y]-1] <= Btemp[Qt*d + Q[X]]) {
				if (Q[Y] < sizeAi && (Q[X] == sizeBi || Atemp[Qt*d + Q[Y]] <= Btemp[Qt*d + Q[X]]) ) {
						M[gbx * d + tidx] = Atemp[Qt*d + Q[Y]];
				}
				else {
						M[gbx * d + tidx] = Btemp[Qt*d + Q[X]];
				}
				// printf("%d\n", M[gbx*d + tidx]);
				break ;
			}
			else {
				K[X] = Q[X] + 1;
				K[Y] = Q[Y] - 1;
			}
		}
		else {
			P[X] = Q[X] - 1;
			P[Y] = Q[Y] + 1 ;
		}
	}
}

int main() {

	// Graine aléatoire
	srand(0);

	// GPU Timer instructions
	float TimerV;
	hipEvent_t start, stop;

	// ==================== //
	//      Parameters      //
	// ==================== //
	const int d = 32;
	int N = 100000;

	int threadsPerBlock = 1024;
	// ===================== //

	int numBlocks = (threadsPerBlock - 1 + N * d) / threadsPerBlock;
	// Allocation de la mémoire
	int * aHost = (int *) malloc(N*d * sizeof(int));
	int * bHost = (int *) malloc(N*d * sizeof(int));
	int * mHost = (int *) malloc(N*d * sizeof(int));

	int * sizeAHost = (int*) malloc(N * sizeof(int));
	int * sizeBHost = (int*) malloc(N * sizeof(int));

	// Remmplissage des tableaux Ai et Bi
	for (int i = 0; i < N; i++){

		// Taille aléatoire du tableau A[i]
		int alea = rand() % d;

		sizeAHost[i] = alea;
		sizeBHost[i] = d - sizeAHost[i];

		// Remplissage des tableaux avec des valeurs croissantes car le tab doit etre trié
		for (int j = 0; j < sizeAHost[i]; j ++){
			aHost[i*d+j] = 2*j;
		}
		for (int j = 0; j < sizeBHost[i]; j ++){
			bHost[i*d+j] = 2*j + 1;
		}
	}

	// Test
	{
		assert( isPowerOfTwo( d ) );
		assert( isPowerOfTwo( threadsPerBlock ) );
		assert( threadsPerBlock <= 1024 );
		assert( d <= 1024 );
		assert( threadsPerBlock % d == 0 );
		assert( isSumOfLengthEqualtoD(sizeAHost, sizeBHost, N, d) );
	}
	

	// Allouer la mémoire globale dans le GPU
	int * aDevice, * bDevice, * mDevice ;
	hipMalloc( (void**) &aDevice, N*d * sizeof(int) );
	hipMalloc( (void**) &bDevice, N*d * sizeof(int) );
	hipMalloc( (void**) &mDevice, N*d * sizeof(int) );
	int * sizeADevice, * sizeBDevice;
	hipMalloc( (void**) &sizeADevice, N * sizeof(int) );
	hipMalloc( (void**) &sizeBDevice, N * sizeof(int) );

	// Copier les tableaux vers le GPU
	hipMemcpy( aDevice, aHost, N*d * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( bDevice, bHost, N*d * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( sizeADevice, sizeAHost, N * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( sizeBDevice, sizeBHost, N * sizeof(int), hipMemcpyHostToDevice );

	// GPU Timer instructions
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Lancer le kernel pour fusionner les tableaux 2 à 2
	mergeSmallBatch_k<<<numBlocks, threadsPerBlock>>>( aDevice, sizeADevice, bDevice, sizeBDevice, mDevice, d, N);

	// GPU timer instructions
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&TimerV, start, stop);
	printf("%d, %d, %f\n", N, d, TimerV);

	// Copier les tableaux du device vers host
	hipMemcpy( mHost, mDevice, N*d * sizeof(int), hipMemcpyDeviceToHost );

	
	// Show result for array (Ai, Bi)
	{
		int i = N-1;
		assert(i < N);

		for (int j = 0; j < d; j++){
			printf("Mhost[%d][%d] = %d\n", i, j, mHost[i*d+j]);
		}
		printf("============================\n");
		printf("Ci-dessus est le tableau M numero i=%d sur les N=%d.\n", i, N);
	        printf("C'est un tableau de taille %d, on a fusioné A et B.\n", d);
		if (sizeAHost[i] != 0)
			printf("A est le tableau de %d nombres PAIRS allant 0 à %d.\n", sizeAHost[i], aHost[d*i+sizeAHost[i]-1]);
		if (sizeBHost[i] != 0)
			printf("B est le tableau de %d nombres IMPAIRS allant 1 à %d.\n", sizeBHost[i], bHost[d*i+sizeBHost[i]-1]);
	}
	

	// Liberer la mémoire
	free(aHost);
	free(bHost);
	free(mHost);
	free(sizeAHost);
	free(sizeBHost);
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(mDevice);
	hipFree(sizeADevice);
	hipFree(sizeBDevice);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}


bool isSumOfLengthEqualtoD(int *A, int *B, int N, int d){
	for (int i = 0; i < N; i ++){
		if (A[i] + B[i] != d){
			return false;
		}
	}
	return true;
}

// https://stackoverflow.com/questions/600293/how-to-check-if-a-number-is-a-power-of-2
bool isPowerOfTwo(int x)
{
	    return (x != 0) && ((x & (x - 1)) == 0);
}

