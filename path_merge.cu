
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define X 0
#define Y 1
#define SIZEA 65536
#define SIZEB 65336

#define N_BLOCKS 64
#define N_THREADS 2

__global__ void mergeBig_k(int *A, int *B, int *M, int *A_idx, int *B_idx){

	// Mémoire shared sur laquelle nous allons travaillé
	__shared__ int A_shared[1024];
	__shared__ int B_shared[1024];

	__shared__ int biaisA;
	__shared__ int biaisB;

	// (endA-startA) : taille de A dans la partition
	// (endB-startB) : taille de B dans la partition
	int startA, endA;
	int startB, endB;
	
	// On récupére les index du début et de la fin de A et B par rapport au tableau global
	if (blockIdx.x == 0){
		startA = 0;
		endA = A_idx[blockIdx.x];
		startB = 0;
		endB = B_idx[blockIdx.x];
	}
	else if (blockIdx.x == N_BLOCKS-1){
		startA = A_idx[blockIdx.x-1];
		endA = SIZEA;
		startB = B_idx[blockIdx.x-1];
		endB = SIZEB;
	}
	else{
		startA = A_idx[blockIdx.x-1];
		endA = A_idx[blockIdx.x];
		startB = B_idx[blockIdx.x-1];
		endB = B_idx[blockIdx.x];
	}

	// Notations de l'article
	// Il y a N élements à fusioner
	// N = SIZEA + SIZEB 
	// Chaque partition contient N/p éléments, chaque bloc traite une partition
	// N / p = (endB-startB) + (endA-startA) = (SIZEA+SIZEB) / N_BLOCKS
	// Si Z est le nombre de threads
	// On va fusioner Z éléments à la fois
	// Donc on a besoin de le faire (N / p) / Z fois
	// On va faire bouger la fenetre glissante (N / p) / Z fois
	int iter_max = (blockDim.x - 1 + (endB-startB) + (endA-startA)) / blockDim.x;
	int iter = 0;

	biaisA = 0;
	biaisB = 0;
	do{
		// Pour synchroniser les biais
		__syncthreads();

		// Chargement des valeurs dans la mémoire shared
		if (startA + biaisA + threadIdx.x < endA){
			A_shared[threadIdx.x] = A[startA + biaisA + threadIdx.x];
		}

		if (startB + biaisB + threadIdx.x < endB){
			B_shared[threadIdx.x] = B[startB + biaisB + threadIdx.x];	
		}

		// Pour synchroniser la mémoire shared
		__syncthreads();

		// Récuperer la taille de la fenetre glissante
		// En général c'est le nombre de threads (blockDim.x), i.e On est dans un carré Z * Z normalement
		// Mais la taille peut être inférieure si il y a moins de blockDim.x éléments à charger
		int sizeAshared = endA-startA - biaisA;
		int sizeBshared = endB-startB - biaisB;
		if (sizeAshared < 0)
			sizeAshared = 0;
		if (sizeAshared > blockDim.x && sizeAshared != 0)
			sizeAshared = blockDim.x;
		if (sizeBshared < 0)
			sizeBshared = 0;
		if (sizeBshared > blockDim.x && sizeBshared != 0)
			sizeBshared = blockDim.x;

		// Binary search
		int i = threadIdx.x;

		if (i < sizeAshared + sizeBshared){
			int K[2];
			int P[2];

			if (i > sizeAshared) {
				K[X] = i - sizeAshared;
				K[Y] = sizeAshared;
				P[X] = sizeAshared;
				P[Y] = i - sizeAshared;
			}
			else {
				K[X] = 0;
				K[Y] = i;
				P[X] = i;
				P[Y] = 0;
			}

			while (1) {
				int offset = (abs(K[Y] - P[Y]))/2;
				int Q[2] = {K[X] + offset, K[Y] - offset};

				if (Q[Y] >= 0 && Q[X] <= sizeBshared && (Q[Y] == sizeAshared || Q[X] == 0 || A_shared[Q[Y]] > B_shared[Q[X]-1])) {
					if (Q[X] == sizeBshared || Q[Y] == 0 || A_shared[Q[Y]-1] <= B_shared[Q[X]]) {
						int idx = startA + startB + i + iter * blockDim.x;
						if (Q[Y] < sizeAshared && (Q[X] == sizeBshared || A_shared[Q[Y]] <= B_shared[Q[X]]) ) {
							M[idx] = A_shared[Q[Y]];
							atomicAdd(&biaisA, 1);	// Biais à incrementer 
						}
						else {
							M[idx] = B_shared[Q[X]];
							atomicAdd(&biaisB, 1); // Biais à incrementer
						}
						//printf("blockIdx.x = %d threadIdx.x = %d idx = %d m = %d biaisA = %d\n", blockIdx.x, threadIdx.x, idx, M[idx], biaisA);
						break ;
					}
					else {
						K[X] = Q[X] + 1;
						K[Y] = Q[Y] - 1;
					}
				}
				else {
					P[X] = Q[X] - 1;
					P[Y] = Q[Y] + 1 ;
				}
			}
		}
		iter = iter + 1;
	} while(iter < iter_max);
}

__global__ void pathBig_k(int *A, int *B, int *M, int *A_idx, int *B_idx){

	// Dans ce kernel, on va simplement chercher N_BLOCKS diagonales
	// de telle sorte que chaque bloc traitera N / N_BLOCKS elements dans le second kernel
	int i = (SIZEA + SIZEB)/N_BLOCKS * (blockIdx.x + 1);
	if (blockIdx.x == N_BLOCKS-1){
		return;
	}

	// Binary search
	int K[2];
	int P[2];

	if (i > SIZEA) {
		K[X] = i - SIZEA;
		K[Y] = SIZEA;
		P[X] = SIZEA;
		P[Y] = i - SIZEA;
	}
	else {
		K[X] = 0;
		K[Y] = i;
		P[X] = i;
		P[Y] = 0;
	}

	while (1) {

		int offset = (abs(K[Y] - P[Y]))/2;
		int Q[2] = {K[X] + offset, K[Y] - offset};

		if (Q[Y] >= 0 && Q[X] <= SIZEB && (Q[Y] == SIZEA || Q[X] == 0 || A[Q[Y]] > B[Q[X]-1])) {
			if (Q[X] == SIZEB || Q[Y] == 0 || A[Q[Y]-1] <= B[Q[X]]) {
				if (Q[Y] < SIZEA && (Q[X] == SIZEB || A[Q[Y]] <= B[Q[X]]) ) {
					M[i] = A[Q[Y]];
				}
				else {
					M[i] = B[Q[X]];
				}
				A_idx[blockIdx.x] = Q[Y];
				B_idx[blockIdx.x] = Q[X];
				// printf("blockIdx.x = %d | Aidx[%d] = %d | Bidx[%d] = %d \n", blockIdx.x, blockIdx.x, Q[Y], blockIdx.x, Q[X]);
				break ;
			}
			else {
				K[X] = Q[X] + 1;
				K[Y] = Q[Y] - 1;
			}
		}
		else {
			P[X] = Q[X] - 1;
			P[Y] = Q[Y] + 1;
		}
	}
}

int main(){

	// Allocation de la mémoire, remplissage du tableau
	int *A = (int*) malloc(sizeof(int) * SIZEA);
	for (int i = 0; i < SIZEA; i++){
		A[i] = 2 * i;
	}
	int *B = (int*) malloc(sizeof(int) * SIZEB);
	for (int i = 0; i < SIZEB; i++){
		B[i] = 2 * i + 1;
	}
	int mHost[SIZEA + SIZEB];		// Tableau merged	

	int A_idx[N_BLOCKS];			// Merge path
	int B_idx[N_BLOCKS];			// Merge path
	int *aDevice, *bDevice, *mDevice, *A_idxDevice, *B_idxDevice;

	// Allocation de la mémoire globale du GPU
	hipMalloc( (void**) &aDevice, SIZEA * sizeof(int) );
	hipMalloc( (void**) &bDevice, SIZEB * sizeof(int) );
	hipMalloc( (void**) &mDevice, (SIZEA+SIZEB) * sizeof(int) );
	hipMalloc( (void**) &A_idxDevice, N_BLOCKS * sizeof(int) );
	hipMalloc( (void**) &B_idxDevice, N_BLOCKS * sizeof(int) );

	// Copier les tableaux vers le GPU
	hipMemcpy( aDevice, A, SIZEA * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( bDevice, B, SIZEB * sizeof(int), hipMemcpyHostToDevice );

	// Lancer le kernel pour trouver une partition des tableaux
	// (SIZEA+SIZEB) / N_BLOCKS elements à traiter pour chaque bloc dans le second kernel
	pathBig_k<<<N_BLOCKS, 1>>>(aDevice, bDevice, mDevice, A_idxDevice, B_idxDevice);

//	cudaMemcpy( mHost, mDevice, (SIZEA+SIZEB) * sizeof(int), cudaMemcpyDeviceToHost );
//	cudaMemcpy( A_idx, A_idxDevice, N_BLOCKS * sizeof(int), cudaMemcpyDeviceToHost );
//	cudaMemcpy( B_idx, B_idxDevice, N_BLOCKS * sizeof(int), cudaMemcpyDeviceToHost );

//	A_idx[N_BLOCKS-1] = SIZEA;
//	B_idx[N_BLOCKS-1] = SIZEB;

//	cudaMemcpy( A_idxDevice, A_idx, N_BLOCKS * sizeof(int), cudaMemcpyHostToDevice );
//	cudaMemcpy( B_idxDevice, B_idx, N_BLOCKS * sizeof(int), cudaMemcpyHostToDevice );

	// (SIZEA+SIZEB) / N_BLOCKS elements à traiter pour chaque bloc dans le second kernel
	// Fenetre glissante pour charger les éléménts dans la mémoire shared
	mergeBig_k<<<N_BLOCKS, N_THREADS>>>(aDevice, bDevice, mDevice, A_idxDevice, B_idxDevice);

	// Copier le tableau résultat vers le CPU, puis affichage
	hipMemcpy( mHost, mDevice, (SIZEA+SIZEB) * sizeof(int), hipMemcpyDeviceToHost );
	for (int i = 0; i < SIZEA+SIZEB; i ++){
		printf("m[%d] = %d\n", i, mHost[i]);
	}

	// Liberation de la mémoire
	free(A);
	free(B);
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(mDevice);
	hipFree(A_idxDevice);
	hipFree(B_idxDevice);

	return 0;
}

